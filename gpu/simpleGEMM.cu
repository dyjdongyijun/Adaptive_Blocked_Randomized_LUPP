#include "util.hpp"

hipblasHandle_t *handle = NULL;

void SimpleGEMM(int n, double *dA, double *dB, double *dC) {
  double alpha = 1.0, beta = 0.0;
  if (!handle) {
    handle = new hipblasHandle_t;
    CHECK_CUBLAS( hipblasCreate(handle) );
  }
  CHECK_CUBLAS( hipblasDgemm(*handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
        n, n, n, &alpha, dA, n, dB, n, &beta, dC, n) );
}

