#include "hip/hip_runtime.h"
#include "rid.hpp"
#include "types.hpp"
#include "gemm.hpp"
#include "random.hpp"
#include "util.hpp"
#include "handle.hpp"
#include "print.hpp"
#include "timer.hpp"
#include "submatrix.hpp"
#include "permute.hpp"


struct PermuteColumn : public thrust::unary_function<int, int> {
  iptr P;
  int  m;

  __host__
    PermuteColumn(iptr P_, int m_):
      P(P_), m(m_)  {}

  __device__
    int operator()(int i) {
      int col = i/m; // after permutation
      int row = i%m;
      return row + P[col] * m;
    }
};


void RandLUPP_OS(const double *A, int m, int n, int k,
    int *&sk, int *&rd, double *&T, double &flops) {

  // over-sampling 
  int r = 2*k;
  assert( r >= k );

  Timer t;

  t.start();
  dvec Gmat(r*m);
  dvec Ymat(r*n);
  if (!sk) CHECK_CUDA( hipFree(sk) );
  if (!rd) CHECK_CUDA( hipFree(rd) );
  if (!T)  CHECK_CUDA( hipFree(T)  );
  CHECK_CUDA( hipMalloc((void **) &sk, sizeof(int)*k) );
  CHECK_CUDA( hipMalloc((void **) &rd, sizeof(int)*(n-k) ));
  CHECK_CUDA( hipMalloc((void **) &T,  sizeof(double)*k*(n-k) ));
  t.stop(); double t4 = t.elapsed_time();
  

  t.start();
  Random::Gaussian(Gmat, 0., 1./std::sqrt(m));
  t.stop(); double t0 = t.elapsed_time();
  
  
  t.start();
  double *G  = thrust::raw_pointer_cast( Gmat.data() );
  double *Yr = thrust::raw_pointer_cast( Ymat.data() );
  GEMM(r, m, n, G, A, Yr); // Y = G * A
  t.stop(); double t1 = t.elapsed_time();
  //print(Ymat, r, n, "Y");

  t.start();
  // Yk = Y(1:k,:)', transpose of the first k rows
  dvec Ykmat(n*k);  double *Yk  = thrust::raw_pointer_cast(Ykmat.data());
  dvec dummat(n*k); double *dum = thrust::raw_pointer_cast(dummat.data()); // dummy variable
  double one = 1.0, zero = 0.0;
  auto const& handle = Handle_t::instance();
  CUBLAS_CHECK( hipblasDgeam( 
        handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
        n, k, 
        &one, Yr, r,
        &zero, dum, n,
        Yk, n) );
  

  dvec work; // working memory for LU
  ivec ipiv( std::min(n,k) ); // local permutation
  ivec info(1); // an integer on gpu

  // query working space
  int lwork = 0;
  CUSOLVER_CHECK( hipsolverDnDgetrf_bufferSize(
        handle.solver, n, k, Yk, n, &lwork) );
  work.resize(lwork);
  
  // compute factorization
  CUSOLVER_CHECK( hipsolverDnDgetrf(
        handle.solver, n, k, 
        Yk, n, 
        thrust::raw_pointer_cast(work.data()), 
        thrust::raw_pointer_cast(ipiv.data()), 
        thrust::raw_pointer_cast(info.data()) ));
  assert( info[0]==0 );
  t.stop(); double t2 = t.elapsed_time();

  // permutation indices
  t.start();
  ivec P(n);
  thrust::sequence(P.begin(), P.end(), 0);
  pivots_to_permutation(ipiv, P);

  // permute columns of Yr
  dvec Ypmat(r*n); double *Yp = thrust::raw_pointer_cast(Ypmat.data());
  {
    auto zero = thrust::make_counting_iterator<int>(0);  
    auto iter = thrust::make_transform_iterator(zero, PermuteColumn(P.data(), r));
    auto elem = thrust::make_permutation_iterator(dptr(Yr), iter);
    thrust::copy_n(elem, r*n, dptr(Yp));    
  }
  t.stop(); double t6 = t.elapsed_time();
  //print( ipiv, "pivots" );
  //print( P, "permutation" );
  //print( Ypmat, r, n, "Yp" );

  
  t.start();
#if 0
  size_t lwork_bytes;
  CUSOLVER_CHECK( hipsolverDnDDgels_bufferSize(
        handle.solver, r, k, n-k,
        NULL, r, 
        NULL, r,
        NULL, k,
        NULL, &lwork_bytes) );
  
  //if (lwork_bytes > 8*work.size()) work.resize(lwork/8.+1);
  //std::cout<<"lwork bytes: "<<lwork_bytes<<std::endl;
  std::cout<<"r: "<<r<<", k: "<<k<<", n: "<<n<<std::endl;
  
  void  *dwork;
  CHECK_CUDA( hipMalloc((void **) &dwork,  lwork_bytes ));

  int niter;
  CUSOLVER_CHECK( hipsolverDnDDgels(
        handle.solver, r, k, n-k,
        Yp, r, 
        Yp+r*k, r,
        T, k,
        dwork, lwork_bytes,
        &niter, 
        thrust::raw_pointer_cast(info.data()) ));
  

  std::cout<<"Info: "<<info[0]<<std::endl;
  std::cout<<"Least squares niters: "<<niter<<std::endl;
  assert( info[0]==0 );
#else
 

#endif
  t.stop(); double t3 = t.elapsed_time();
  


  t.start();
  thrust::copy_n( P.begin(), k, iptr(sk) );
  thrust::copy_n( P.begin()+k, n-k, iptr(rd) );
  
  /*
  auto ZERO = thrust::make_counting_iterator<int>(0);
  auto indx = thrust::make_transform_iterator(ZERO, SubMatrix(n-k, n));
  auto elem = thrust::make_permutation_iterator(dptr(Yk+k), indx);
  thrust::copy_n( elem, (n-k)*k, dptr(dum) ); // dummy array n-by-k

  // transpose
  CUBLAS_CHECK( hipblasDgeam(
        handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
        k, n-k,
        &one, dum, n-k,
        &zero, T, k,
        T, k) );
  */
  t.stop(); double t5 = t.elapsed_time();


  flops = 2.*m*n*k + 2.*m*k*k/3. + 1.0*k*k*(m-k);


#ifdef PROF
  std::cout<<std::endl
    <<"--------------------\n"
    <<"  RandLUPP-OS\n"
    <<"--------------------\n"
    <<"Alloc: "<<t4<<std::endl
    <<"Rand:  "<<t0<<std::endl
    <<"GEMM:  "<<t1<<std::endl
    <<"LUPP:  "<<t2<<std::endl
    <<"Solve: "<<t3<<std::endl
    <<"Copy:  "<<t5<<std::endl
    <<"Perm:  "<<t6<<std::endl
    <<"--------------------\n"
    <<"Total: "<<t0+t1+t2+t3+t4+t5+t6<<std::endl
    <<"--------------------\n"
    <<std::endl;
#endif
}

